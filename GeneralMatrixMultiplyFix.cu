#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

typedef struct{
	int width;
	int height;
	int* elements;
} Matrix;

__global__ void matrixProduct(Matrix, Matrix, Matrix, Matrix, Matrix, int*, int);

int matrixProduct(Matrix, Matrix, int, int);
void printMatrix(Matrix, char[]);

main()
{
	//Declare vars, constants
	int const MatSize=999;
	Matrix Matrix1, Matrix2, Result, Res_Check, BlockRow, BlockCol;
	Matrix dev_Matrix1, dev_Matrix2, dev_Result, dev_BlockRow, dev_BlockCol;
	
	//Debug Code
	BlockRow.height = MatSize; BlockRow.width = MatSize;
	BlockCol.height = MatSize; BlockCol.width = MatSize;
	
	//For generalization purposes
	int sections, numThreads;
	int *startPoint, *dev_startPoint;
	numThreads = 100;
	sections=(((MatSize+1)*(MatSize+1))/numThreads);
	
	Matrix1.width = MatSize; Matrix1.height = MatSize;
	Matrix2.width = MatSize; Matrix2.height = MatSize;
	Result.width = MatSize; Result.height = MatSize;
	Res_Check.width = MatSize; Res_Check.height = MatSize;
	 
	dim3 blockSize= numThreads; //make a linear allocation of threads to compute the matrix multiply
	dim3 gridSize(1,1);

	int i,j;
	struct timeval start, elapsed, end, err_start, err_end, err_elapsed;

	size_t MemSize = (MatSize+1) * (MatSize+1) * sizeof(int);
	Matrix1.elements = (int*) malloc(MemSize);
	Matrix2.elements = (int*) malloc(MemSize);
	Result.elements = (int*) malloc(MemSize);
	Res_Check.elements = (int*) malloc(MemSize);
	startPoint = (int*) malloc(sections*sizeof(int));
	
	BlockRow.elements = (int*) malloc(MemSize);
	BlockCol.elements = (int*) malloc(MemSize);
	
	//Initialize matrices with random values
	for(i=0;i<=MatSize;i++)
	{
		for(j=0;j<=MatSize;j++)
		{
			Matrix1.elements[(i*(Matrix1.width+1))+j]=(i*(Matrix1.width+1))+j;
			Matrix2.elements[(i*(Matrix1.width+1))+j]=(i*(Matrix1.width+1))+j;
		}
	}
	
	//Fill out the array of starting points in the matrix
	startPoint[0]=0;
	for(i=1;i<sections;i++)
	{
		startPoint[i]=startPoint[i-1]+sections;
		//printf("Starting element for thread %d is: %d \n", i,startPoint[i]);
	}

	gettimeofday(&start,NULL);
	printf("Start Values %ld, %ld\n",start.tv_sec,start.tv_usec);
	
	//Transfer matrices to device memory
	
	dev_Matrix1.height = Matrix1.height; dev_Matrix1.width = Matrix1.width;
	hipMalloc((void**)&dev_Matrix1.elements,MemSize);
	hipMemcpy(dev_Matrix1.elements, Matrix1.elements, MemSize, hipMemcpyHostToDevice);

	dev_Matrix2.height = Matrix2.height; dev_Matrix2.width = Matrix2.width;
	hipMalloc((void**)&dev_Matrix2.elements,MemSize);
	hipMemcpy(dev_Matrix2.elements, Matrix2.elements, MemSize, hipMemcpyHostToDevice);
	
	hipMalloc((void**)&dev_startPoint,sections*sizeof(int));
	hipMemcpy(dev_startPoint,startPoint,(sections*sizeof(int)),hipMemcpyHostToDevice);
	
	dev_BlockRow.height = BlockRow.height; dev_BlockRow.width = BlockRow.width;
	dev_BlockCol.height = BlockCol.height; dev_BlockCol.width = BlockCol.width;
	hipMalloc((void**)&dev_BlockRow.elements,MemSize);
	hipMalloc((void**)&dev_BlockCol.elements,MemSize);
	
	dev_Result.height = Result.height; dev_Result.width = Result.width;
	hipMalloc((void**)&dev_Result.elements,MemSize);
	
		
	//Kernel Declaration
	matrixProduct<<<blockSize,gridSize>>>(dev_Matrix1, dev_Matrix2, dev_Result, dev_BlockRow, dev_BlockCol, dev_startPoint, sections);
	hipMemcpy(Result.elements, dev_Result.elements, MemSize, hipMemcpyDeviceToHost);
	hipMemcpy(BlockRow.elements, dev_BlockRow.elements, MemSize, hipMemcpyDeviceToHost);
	hipMemcpy(BlockCol.elements, dev_BlockCol.elements, MemSize, hipMemcpyDeviceToHost);
	
	gettimeofday(&end,NULL);
	//printf("End Values %ld, %ld\n",end.tv_sec,end.tv_usec);
	
	//printMatrix(Matrix1,"Matrix 1\n");
	//printMatrix(Matrix2, "Matrix 2\n");
	//printMatrix(Result, "Result Matrix\n");
	//printMatrix(BlockRow, "Compute Row Used\n");
	//printMatrix(BlockCol, "Compute Column Used\n");
	
	
	//Compute time elapsed
	
	elapsed.tv_sec = (end.tv_sec-start.tv_sec);
	elapsed.tv_usec = (((elapsed.tv_sec*1000000)+end.tv_usec)-start.tv_usec);
	printf("Elapsed Time: %ld \n",(elapsed.tv_usec));
	
	//Check the output for errors
	for(i=0;i<=MatSize;i++)
	{
		for(j=0;j<=MatSize;j++)
		{
			Res_Check.elements[(i*(Res_Check.width+1))+j] = matrixProduct(Matrix1, Matrix2, i, j);
			if(Res_Check.elements[(i*(Res_Check.width+1))+j] != Result.elements[(i*(Result.width+1))+j])
			{
				printf("Error found in row %d, column %d\n",i,j);
				printf("Value in parallel: %d, Value in host comp: %d\n",Result.elements[(i*(Result.width+1))+j],Res_Check.elements[(i*(Res_Check.width+1))+j]);
			}
		}
	}
	//printMatrix(Res_Check,"Error-Check Matrix\n");
	printf("Error Check finished\n");
}

int matrixProduct(Matrix Mat1, Matrix Mat2,int row, int col)
{
	int k,sum;
	sum=0;
	for(k=0;k<=Mat1.width;k++)
	{	
		sum=sum+(Mat1.elements[(row*(Mat1.width+1))+k])*(Mat2.elements[(k*(Mat2.width+1))+col]);	
	}
	return sum;
}

void printMatrix(Matrix Mat, char name[])
{
	int i,j;
	printf(name);
	for(i=0;i<=Mat.width;i++)
		{
			for(j=0;j<=Mat.width;j++)
			{
				printf("%d\t",Mat.elements[(i*(Mat.width+1))+j]);
			}
			printf("\n");
		}
	return;
}

__global__ void matrixProduct(Matrix Mat1, Matrix Mat2, Matrix Res, Matrix bkRow, Matrix bkCol, int* start, int threadSize)
{
	int thread = blockIdx.x;
	int k,sum,index,row,col;
	
	for(index=start[thread];index<(start[thread]+threadSize);index++)
	{
		sum=0;
		row = index / (Mat1.width+1);
		col = index % (Mat1.width+1);
		for(k=0;k<=Mat1.width;k++)
		{
			sum=sum+((Mat1.elements[(row*(Mat1.width+1))+k])*(Mat2.elements[(k*(Mat2.width+1))+col]));
		}
		Res.elements[index]=sum;
		bkRow.elements[index] = row;
		bkCol.elements[index] = col;
	}
	
	/*int row = blockIdx.x;
	int col = blockIdx.y;
	int k,sum;
	sum=0;
	for(k=0;k<=Mat1.width;k++)
	{
		sum=sum+(Mat1.elements[(row*(Mat1.width+1))+k])*(Mat2.elements[(k*(Mat2.width+1))+col]);
	}
	Res.elements[(row*(Res.width+1))+col]=sum;*/
	
}
