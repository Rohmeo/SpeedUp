#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

typedef struct{
	int width;
	int height;
	int* elements;
} Matrix;

__global__ void matrixProduct(Matrix, Matrix, Matrix, Matrix, Matrix);

int matrixProduct(Matrix, Matrix, int, int);
void printMatrix(Matrix, char[]);

main()
{
	//Declare vars, constants
	int const MatSize=3;
	Matrix Matrix1, Matrix2, Result, Res_Check, BlockRow, BlockCol;
	Matrix dev_Matrix1, dev_Matrix2, dev_Result, dev_BlockRow, dev_BlockCol;
	
	//Code to aid in debugging the non-determinism
	BlockRow.height = MatSize; BlockRow.width = MatSize;
	BlockCol.height = MatSize; BlockCol.width = MatSize;

	Matrix1.width = MatSize; Matrix1.height = MatSize;
	Matrix2.width = MatSize; Matrix2.height = MatSize;
	Result.width = MatSize; Result.height = MatSize;
	Res_Check.width = MatSize; Res_Check.height = MatSize;
	 
	dim3 blockSize(MatSize+1,MatSize+1);
	dim3 gridSize(1,1);

	int i,j;
	struct timeval start, elapsed, end;

	size_t MemSize = (MatSize+1) * (MatSize+1) * sizeof(int);
	Matrix1.elements = (int*) malloc(MemSize);
	Matrix2.elements = (int*) malloc(MemSize);
	Result.elements = (int*) malloc(MemSize);
	Res_Check.elements = (int*) malloc(MemSize);
	
	BlockRow.elements = (int*) malloc(MemSize);
	BlockCol.elements = (int*) malloc(MemSize);
	
	//Initialize matrices with random values
	for(i=1;i<=MatSize;i++)
	{
		for(j=1;j<=MatSize;j++)
		{
			Matrix1.elements[(i*Matrix1.width)+j]=(i*Matrix1.width)+j;
			Matrix2.elements[(i*Matrix1.width)+j]=(i*Matrix1.width)+j;
		}
	}
	gettimeofday(&start,NULL);
	printf("Start Values %ld, %ld\n",start.tv_sec,start.tv_usec);
	
	//Transfer matrices to device memory
	
	dev_Matrix1.height = Matrix1.height; dev_Matrix1.width = Matrix1.width;
	hipMalloc((void**)&dev_Matrix1.elements,MemSize);
	hipMemcpy(dev_Matrix1.elements, Matrix1.elements, MemSize, hipMemcpyHostToDevice);

	dev_Matrix2.height = Matrix2.height; dev_Matrix2.width = Matrix2.width;
	hipMalloc((void**)&dev_Matrix2.elements,MemSize);
	hipMemcpy(dev_Matrix2.elements, Matrix2.elements, MemSize, hipMemcpyHostToDevice);
	
	dev_Result.height = Result.height; dev_Result.width = Result.width;
	hipMalloc((void**)&dev_Result.elements,MemSize);
	
	dev_BlockRow.height = BlockRow.height; dev_BlockRow.width = BlockRow.width;
	dev_BlockCol.height = BlockCol.height; dev_BlockCol.width = BlockCol.width;
	hipMalloc((void**)&dev_BlockRow.elements,MemSize);
	hipMalloc((void**)&dev_BlockCol.elements,MemSize);
		
	//Kernel Declaration
	matrixProduct<<<blockSize,gridSize>>>(dev_Matrix1, dev_Matrix2, dev_Result, dev_BlockRow, dev_BlockCol);
	
	hipMemcpy(Result.elements, dev_Result.elements, MemSize, hipMemcpyDeviceToHost);
	hipMemcpy(BlockRow.elements, dev_BlockRow.elements, MemSize, hipMemcpyDeviceToHost);
	hipMemcpy(BlockCol.elements, dev_BlockCol.elements, MemSize, hipMemcpyDeviceToHost);
	
	gettimeofday(&end,NULL);
	printf("End Values %ld, %ld\n",end.tv_sec,end.tv_usec);
	
	printMatrix(Matrix1,"Matrix 1\n");
	printMatrix(Matrix2, "Matrix 2\n");
	printMatrix(Result, "Result Matrix\n");
	printMatrix(BlockRow, "Compute Row Used\n");
	printMatrix(BlockCol, "Compute Column Used\n");
	
	
	//Compute time elapsed
	
	elapsed.tv_sec = (end.tv_sec-start.tv_sec);
	if(end.tv_usec > start.tv_usec)
	{
		elapsed.tv_usec = (end.tv_usec-start.tv_usec);
	}
	else
	{
		elapsed.tv_usec = (((elapsed.tv_sec*1000000)+end.tv_usec)-start.tv_usec);
	}
	printf("Elapsed Time: %ld \n",/*((elapsed.tv_sec)*1000000)+*/(elapsed.tv_usec));
	
	//Check the output for errors
	for(i=1;i<=MatSize;i++)
	{
		for(j=1;j<=MatSize;j++)
		{
			Res_Check.elements[(i*Res_Check.width)+j] = matrixProduct(Matrix1, Matrix2, i, j);
			if(Res_Check.elements[(i*Res_Check.width)+j] != Result.elements[(i*Result.width)+j])
			{
				printf("Error found in row %d, column %d\n",i,j);
				printf("Value in parallel: %d, Value in host comp: %d\n",Result.elements[(i*Result.width)+j],Res_Check.elements[(i*Res_Check.width)+j]);
			}
		}
	}
	printMatrix(Res_Check,"Error-Check Matrix\n");
	printf("Error Check finished\n");
}

int matrixProduct(Matrix Mat1, Matrix Mat2,int row, int col)
{
	int k,sum;
	sum=0;
	for(k=1;k<=Mat1.width;k++)
	{	
		sum=sum+(Mat1.elements[(row*Mat1.width)+k])*(Mat2.elements[(k*Mat2.width)+col]);	
	}
	return sum;
}

void printMatrix(Matrix Mat, char name[])
{
	int i,j;
	printf(name);
	for(i=1;i<=Mat.width;i++)
		{
			for(j=1;j<=Mat.width;j++)
			{
				printf("%d\t",Mat.elements[(i*Mat.width)+j]);
			}
			printf("\n");
		}
	return;
}

__global__ void matrixProduct(Matrix Mat1, Matrix Mat2, Matrix Res, Matrix bkRow, Matrix bkCol)
{
	int row = blockIdx.x;
	int col = blockIdx.y;
	int k,sum;
	sum=0;
	for(k=1;k<=Mat1.width;k++)
	{
		sum=sum+(Mat1.elements[(row*Mat1.width)+k])*(Mat2.elements[(k*Mat2.width)+col]);
	}
	bkRow.elements[(row*bkRow.width)+col]=row;
	bkCol.elements[(row*bkRow.width)+col]=col;
	Res.elements[(row*Res.width)+col]=sum;
}
