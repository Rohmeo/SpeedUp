#include "hip/hip_runtime.h"
#import <stdlib.h>
#import <sys/time.h>
#import <stdio.h>

__global__ void passCrack(int,char*,char*, char*, char*); //variables are length, password, list of divisions within the guessing area, answer, and starting guesses

__device__ char strTest(char*, char*);

main(){
	int i,thread;
	int const length = 2;
	char* pass, *answer, *division, *start;
	int const numThreads = 3;
	char const range = 75;
	char *dev_pass, *dev_answer, *dev_division, *dev_start;
	
	//Create space on host (password, division, answer need allocating)
	size_t strSize = length*sizeof(char);
	pass = (char*) malloc(strSize);
	answer = (char*) malloc(strSize);
	division = (char*) malloc(numThreads*sizeof(char));
	start = (char*) malloc(numThreads*length*sizeof(char));
	pass = "Hi";
	
	
	//prepare the 'division' variable
	for(i=0;i<numThreads;i++){
		division[i] = range/i;
		for(j=0;j<length;j++){		//this loop prepares the starting values guessing (first two to '0', 2nd pair to '0'+25, etc.)
			start[(i*length)+j] = '0'+division[i]
		}
	}

	//Create space on device, copy to device (password, division, start need to be copied)
	hipMalloc((void**)&dev_pass, strSize);
	hipMemcpy(dev_pass, pass, strSize, hipMemcpyHostToDevice);
	
	hipMalloc((void**)&dev_start, (numThreads*length*sizeof(char));
	hipMemcpy(dev_start, start, (numThreads*length*sizeof(char)), hipMemcpyHostToDevice);	

	hipMalloc((void**)&dev_answer, strSize);
	hipMalloc((void**)&dev_division,(numThreads*sizeof(char)));
	hipMemcpy(dev_division, division, (numThreads*sizeof(char)),hipMemcpyHostToDevice);
	
	//Initialize the Kernel
	dim3 blockSize = numThreads;
	passCrack<<<blockSize, 1>>>(length, dev_pass, dev_division, dev_answer, dev_start);
	//Copy result from device to host
	hipMemcpy(answer, dev_answer, strSize, hipMemcpyDeviceToHost);
	printf("The password is '%c%c'",answer[0],answer[1]);
	//timing
	
}

__global__ void passCrack(int length, char* pass, char* division, char* answer, char* start){
	int thread = blockIdx.x;
	int const startArrayLength = length*blockDim.x;
	char start[startArrayLength];
	int i,j,k,cpy;
	
	for(i=0;i<(75/blockDim.x);i++){
		start[(thread*length)+1]='0'+division[thread];  
		for(j=0;j<(75/blockDim.x);j++){
			if(strTest(&pass[0],&start[thread*length]) == 1){
				//This code is written to run on each core, but will only execute once, on the one where the password is matched
				for(cpy=0;cpy<length;cpy++){
					answer[i]=start[(thread*length)+i];
				}
			start[(thread*length)+1]++;
			}
		}
		start[thread*length]++;
	}
}

__device__ char strTest(char* pass, char* guess){
	if(guess[0]==pass[0]){
		if(guess[1]==pass[1]){
			return 1;
		}
		else return 0;
	}
	else return 0;
}
