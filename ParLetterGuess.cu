#include "hip/hip_runtime.h"
#import <stdlib.h>
#import <sys/time.h>
#import <stdio.h>

__global__ void passCrack(int, int*, char*,char*, char*, char*); //variables are length, password, list of divisions within the guessing area, answer, and starting guesses

__device__ char strTest(char*, char*, int*);

main(){
	int i,j;
	int const length = 2;
	char *newpass, *pass, *answer, *division, *start;
	int const numThreads = 3;
	char const range = 75;
	char *dev_pass, *dev_answer, *dev_division, *dev_start;
	
	int *debug, *dev_debug;
	
	//Create space on host (password, division, answer need allocating)
	size_t strSize = length*sizeof(char);
	
	debug = (int*) malloc(numThreads*sizeof(int));
	pass = (char*) malloc(strSize);
	newpass = (char*) malloc(strSize);
	answer = (char*) malloc(strSize);
	division = (char*) malloc(numThreads*sizeof(char));
	start = (char*) malloc(numThreads*length*sizeof(char));
	pass = "Hi";
	
	
	//prepare the 'division' variable
	//for(i=0;i<numThreads;i++){
	//	if(i==0){division[i]=0;}	
	//	else {division[i] = range/i;}
	//	for(j=0;j<length;j++){		//this loop prepares the starting values guessing (first two to '0', 2nd pair to '0'+25, etc.)
	//		start[(i*length)+j] = '0'+division[i];
	//	}
	//}

	//Create space on device, copy to device (password, division, start need to be copied)
	hipMalloc((void**)&dev_pass, strSize);
	hipMemcpy(dev_pass, pass, strSize, hipMemcpyHostToDevice);
	
	hipMalloc((void**)&dev_start, (numThreads*length*sizeof(char)));
	hipMemcpy(dev_start, start, (numThreads*length*sizeof(char)), hipMemcpyHostToDevice);	
	
	hipMalloc((void**)&dev_debug, (numThreads*sizeof(int)));

	hipMalloc((void**)&dev_answer, strSize);
	
	hipMalloc((void**)&dev_division,(numThreads*sizeof(char)));
	hipMemcpy(dev_division, division, (numThreads*sizeof(char)),hipMemcpyHostToDevice);
	
	//Initialize the Kernel
	dim3 blockSize = numThreads;
	printf("Start %c%c%c%c%c%c\n",start[0],start[1],start[2],start[3],start[4],start[5]);
	passCrack<<<blockSize, 1>>>(length, dev_debug, dev_pass, dev_division, dev_answer, dev_start);
	
	//Copy result from device to host
	hipMemcpy(answer, dev_answer, strSize, hipMemcpyDeviceToHost);
	hipMemcpy(start,dev_start, (numThreads*length*sizeof(char)), hipMemcpyDeviceToHost);
	
	hipMemcpy(debug, dev_debug, (numThreads*sizeof(int)), hipMemcpyDeviceToHost);
	
	printf("The password is '%c%c'\n",answer[0],answer[1]);
	printf("Start %c%c%c%c%c%c\n",start[0],start[1],start[2],start[3],start[4],start[5]);
	//timing
	printf("Threads: %d\t%d\t%d\n",debug[0],debug[1],debug[2]);
}

__global__ void passCrack(int length, int* debug, char* pass, char* division, char* answer, char* start){
	int thread = blockIdx.x;
	int top = 75/blockDim.x;
	int i,j,k,cpy;
	start[0] = '0';
	start[1] = '0';	
	debug[thread] = thread;
	
	for(start[0]='0';start[0]<'{';start[0]++){
		start[1]='0';  
		for(start[1]='0';start[1]<'{';start[1]++){
			if(strTest(&pass[0],&start[0],&debug[0]) == 1){
				debug[1] = 9;
				//This code is written to run on each core, but will only execute once, on the one where the password is matched
				for(cpy=0;cpy<length;cpy++){
					answer[cpy]=start[cpy];
				}
			}
		}
	}
}

__device__ char strTest(char* pass, char* guess,int* debug){
	if(guess[0]==pass[0]){
		if(guess[1]==pass[1]){
			debug[2] = 10;
			return 1;
		}
		else{
			debug[1]=10;
			return 0;
		}
	}

	else{ 
		debug[0] = 10;
		return 0;
	}
}
